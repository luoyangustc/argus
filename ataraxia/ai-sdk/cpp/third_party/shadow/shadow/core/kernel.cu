#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include "util/log.hpp"

#include <cmath>

namespace Shadow {

namespace Kernel {

#if defined(USE_CUDA)
hipblasHandle_t cublas_handle_ = nullptr;
#if defined(USE_CUDNN)
hipdnnHandle_t cudnn_handle_ = nullptr;
#endif

void Setup(int device_id) {
  if (cublas_handle_ == nullptr) {
    CUDA_CHECK(hipSetDevice(device_id));
    hipblasCreate(&cublas_handle_);
    CHECK_NOTNULL(cublas_handle_);
  }

#if defined(USE_CUDNN)
  if (cudnn_handle_ == nullptr) {
    CUDNN_CHECK(hipdnnCreate(&cudnn_handle_));
    CHECK_NOTNULL(cudnn_handle_);
  }
#endif
}

void Release() {
  if (cublas_handle_ != nullptr) {
    hipblasDestroy(cublas_handle_);
    cublas_handle_ = nullptr;
  }

#if defined(USE_CUDNN)
  if (cudnn_handle_ != nullptr) {
    CUDNN_CHECK(hipdnnDestroy(cudnn_handle_));
    cudnn_handle_ = nullptr;
  }
#endif
}

void Synchronize() { CUDA_CHECK(hipDeviceSynchronize()); }

template <typename T, typename Dtype>
T *MakeBuffer(int size, Dtype *host_ptr) {
  T *buffer;
  CUDA_CHECK(hipMalloc(&buffer, size * sizeof(Dtype)));
  if (host_ptr != nullptr) {
    WriteBuffer(size, host_ptr, buffer);
  }
  return buffer;
}

template <typename T, typename Dtype>
void ReadBuffer(int size, const T *src, Dtype *des) {
  CUDA_CHECK(
      hipMemcpy(des, src, size * sizeof(Dtype), hipMemcpyDeviceToHost));
}

template <typename T, typename Dtype>
void WriteBuffer(int size, const Dtype *src, T *des) {
  CUDA_CHECK(
      hipMemcpy(des, src, size * sizeof(Dtype), hipMemcpyHostToDevice));
}

template <typename T, typename Dtype>
void CopyBuffer(int size, const T *src, T *des) {
  CUDA_CHECK(
      hipMemcpy(des, src, size * sizeof(Dtype), hipMemcpyDeviceToDevice));
}

template <typename T>
void ReleaseBuffer(T *buffer) {
  CUDA_CHECK(hipFree(buffer));
}

// Explicit instantiation
template int *MakeBuffer<int, int>(int size, int *host_ptr);
template float *MakeBuffer<float, float>(int size, float *host_ptr);
template unsigned char *MakeBuffer<unsigned char, unsigned char>(
    int size, unsigned char *host_ptr);

template void ReadBuffer<int, int>(int size, const int *src, int *des);
template void ReadBuffer<float, float>(int size, const float *src, float *des);
template void ReadBuffer<unsigned char, unsigned char>(int size,
                                                       const unsigned char *src,
                                                       unsigned char *des);

template void WriteBuffer<int, int>(int size, const int *src, int *des);
template void WriteBuffer<float, float>(int size, const float *src, float *des);
template void WriteBuffer<unsigned char, unsigned char>(
    int size, const unsigned char *src, unsigned char *des);

template void CopyBuffer<int, int>(int size, const int *src, int *des);
template void CopyBuffer<float, float>(int size, const float *src, float *des);
template void CopyBuffer<unsigned char, unsigned char>(int size,
                                                       const unsigned char *src,
                                                       unsigned char *des);

template void ReleaseBuffer<int>(int *buffer);
template void ReleaseBuffer<float>(float *buffer);
template void ReleaseBuffer<unsigned char>(unsigned char *buffer);
#endif

}  // namespace Kernel

}  // namespace Shadow
