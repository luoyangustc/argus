#include "hip/hip_runtime.h"
#include "roi_pooling_op.hpp"

namespace Shadow {

namespace Vision {

#if defined(USE_CUDA)
template <typename T>
__global__ void KernelPOIPooling(const T *in_data, int count, const T *roi_data,
                                 int in_c, int in_h, int in_w, int pooled_h,
                                 int pooled_w, float spatial_scale,
                                 T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    int pw = globalid % pooled_w;
    int ph = (globalid / pooled_w) % pooled_h;
    int c = (globalid / pooled_w / pooled_h) % in_c;
    int n = globalid / pooled_w / pooled_h / in_c;

    roi_data += n * 5;
    int roi_batch_id = static_cast<int>(roi_data[0]);
    int roi_start_w = static_cast<int>(round(roi_data[1] * spatial_scale));
    int roi_start_h = static_cast<int>(round(roi_data[2] * spatial_scale));
    int roi_end_w = static_cast<int>(round(roi_data[3] * spatial_scale));
    int roi_end_h = static_cast<int>(round(roi_data[4] * spatial_scale));

    int roi_height = max(roi_end_h - roi_start_h + 1, 1);
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);
    T bin_size_h = roi_height / static_cast<T>(pooled_h);
    T bin_size_w = roi_width / static_cast<T>(pooled_w);

    int hstart = static_cast<int>(floor(ph * bin_size_h));
    int wstart = static_cast<int>(floor(pw * bin_size_w));
    int hend = static_cast<int>(ceil((ph + 1) * bin_size_h));
    int wend = static_cast<int>(ceil((pw + 1) * bin_size_w));

    hstart = min(max(hstart + roi_start_h, 0), in_h);
    hend = min(max(hend + roi_start_h, 0), in_h);
    wstart = min(max(wstart + roi_start_w, 0), in_w);
    wend = min(max(wend + roi_start_w, 0), in_w);

    bool is_empty = (hend <= hstart) || (wend <= wstart);

    in_data += (roi_batch_id * in_c + c) * in_h * in_w;

    T max_val = is_empty ? 0 : in_data[hstart * in_w + wstart];
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        max_val = max(max_val, in_data[h * in_w + w]);
      }
    }
    out_data[globalid] = max_val;
  }
}

template <typename T>
void ROIPooling(const T *in_data, const VecInt &in_shape, const T *roi_data,
                int num_rois, int pooled_h, int pooled_w, float spatial_scale,
                T *out_data) {
  int in_c = in_shape[1], in_h = in_shape[2], in_w = in_shape[3];
  int count = num_rois * in_c * pooled_h * pooled_w;
  KernelPOIPooling<T><<<GetBlocks(count), NumThreads>>>(
      in_data, count, roi_data, in_c, in_h, in_w, pooled_h, pooled_w,
      spatial_scale, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template void ROIPooling(const float *in_data, const VecInt &in_shape,
                         const float *roi_data, int num_rois, int pooled_h,
                         int pooled_w, float spatial_scale, float *out_data);
#endif

}  // namespace Vision

}  // namespace Shadow