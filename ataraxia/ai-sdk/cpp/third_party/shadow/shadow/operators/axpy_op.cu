#include "hip/hip_runtime.h"
#include "axpy_op.hpp"

namespace Shadow {

namespace Vision {

#if defined(USE_CUDA)
template <typename T>
__global__ void KernelAxpy(int count, int spatial_dim, const T *scale_data,
                           const T *x_data, const T *y_data, T *out_data) {
  CUDA_KERNEL_LOOP(globalid, count) {
    out_data[globalid] = scale_data[globalid / spatial_dim] * x_data[globalid] +
                         y_data[globalid];
  }
}

template <typename T>
void Axpy(const T *scale_data, const T *x_data, const T *y_data,
          const VecInt &in_shape, T *out_data) {
  int spatial_dim = in_shape[2] * in_shape[3];
  int count = in_shape[0] * in_shape[1] * spatial_dim;
  KernelAxpy<T><<<GetBlocks(count), NumThreads>>>(
      count, spatial_dim, scale_data, x_data, y_data, out_data);
  CUDA_CHECK(hipPeekAtLastError());
}

template void Axpy(const float *scale_data, const float *x_data,
                   const float *y_data, const VecInt &in_shape,
                   float *out_data);
#endif

}  // namespace Vision

}  // namespace Shadow
