#include "hip/hip_runtime.h"
#include "PreluPlugin.hpp"
#include "MathFunction.hpp"
namespace Shadow
{

__global__ void PReLU(const int n, const int channels, const int dim,
    const float* in, float* out, const float* slope_data, const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : in[index] * slope_data[c];
  }
}


void PreluLayer(const int count, const int channels, const int dim, const float* bottom_data,
  float* top_data, void* mDeviceKernel, const int div_factor){
  PReLU<<<GET_BLOCKS(count), CUDA_NUM_THREADS>>>(count,channels,dim,bottom_data,top_data,static_cast<const float*>(mDeviceKernel),div_factor);
}

}//namespace Shadow
